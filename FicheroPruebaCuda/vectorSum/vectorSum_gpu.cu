#include "hip/hip_runtime.h"
// Suma de vectores con kernel en 1 dimensión

#include "common.h"

#define N   10

__global__ void add_block( int *a, int *b, int *c ) {
    int tid = blockIdx.x;    // this thread handles the data at its thread id

    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

__global__ void add_thread( int *a, int *b, int *c ) {
    int tid = threadIdx.x;    // this thread handles the data at its thread id

    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

__global__ void add( int *a, int *b, int *c ) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;    // this thread handles the data at its thread id

    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

int main( void ) {
    int a[N], b[N], c[N], d[N], e[N];
    int *dev_a, *dev_b, *dev_c, *dev_d, *dev_e;

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_c, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_d, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_e, N * sizeof(int) ) );

    // fill the arrays 'a' and 'b' on the CPU
    for (int i=0; i<N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

//----------------------------------------------------------------
    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(int),
                              hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_b, b, N * sizeof(int),
                              hipMemcpyHostToDevice ) );

    add_thread<<<1,N>>>( dev_a, dev_b, dev_c );

    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( c, dev_c, N * sizeof(int),
                              hipMemcpyDeviceToHost ) );

    // display the results
    for (int i=0; i<N; i++) {
        printf( "%d + %d = %d\n", a[i], b[i], c[i] );
    }

//----------------------------------------------------------------
    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(int),
                              hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_b, b, N * sizeof(int),
                              hipMemcpyHostToDevice ) );

    add_block<<<N,1>>>( dev_a, dev_b, dev_d );

    // copy the array 'd' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( d, dev_d, N * sizeof(int),
                              hipMemcpyDeviceToHost ) );

    // display the results
    for (int i=0; i<N; i++) {
        printf( "%d + %d = %d\n", a[i], b[i], d[i] );
    }

//----------------------------------------------------------------
    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(int),
                              hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_b, b, N * sizeof(int),
                              hipMemcpyHostToDevice ) );

    add<<<ceil(N/5.0),5>>>( dev_a, dev_b, dev_e );

    // copy the array 'd' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( e, dev_e, N * sizeof(int),
                              hipMemcpyDeviceToHost ) );

    // display the results
    for (int i=0; i<N; i++) {
        printf( "%d + %d = %d\n", a[i], b[i], e[i] );
    }

//----------------------------------------------------------------

    // free the memory allocated on the GPU
    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipFree( dev_b ) );
    HANDLE_ERROR( hipFree( dev_c ) );
    HANDLE_ERROR( hipFree( dev_d ) );
    HANDLE_ERROR( hipFree( dev_e ) );

    return 0;
}
