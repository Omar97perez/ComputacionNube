#include "hip/hip_runtime.h"
/* ************************************************************************
 * Program: matrix.c
 * Description: Matrix multiplication
 **************************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include "common.h"
#include "matrix_common.h"

#define DEFAULT_N   1
#define BLOCK_SIZE 16

/* ************************************************************************
 * Routine: matmul_kernel 
 * Description: Matrix multiplication
 **************************************************************************/
__global__ void matmul_kernel(float *C, float *A, float *B, int N) {
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;

  if((i<N) && (j<N))
  {
    C[i*N+j] = 0;
    for(int k = 0; k < N; k++)
      C[i*N+j] += A[i*N+k] * B[k*N+j];
  }
}

/* ************************************************************************
 * Routine: main
 * Description: Performs several tests.
 **************************************************************************/
int main(int argc, char *argv[]) {
  int N, sizeinBytes;
  float *A, *B, *C, *D;
  float *d_A, *d_B, *d_C;

  // Timers
  hipEvent_t start, stop;

  /* Command line parameters processing */
  switch(argc) {
    case 1: N = DEFAULT_N;
            break;
    case 2: N = atoi(argv[1]);
            break;
    default: 
            printf("\nUse: %s <N>", argv[0]);
            printf("\nN: Dimensión del GRID en bloques de %d)\n", BLOCK_SIZE);
            break;
  }

  sizeinBytes = N*N*BLOCK_SIZE*BLOCK_SIZE*sizeof(float);

  A = (float*)malloc(sizeinBytes);
  B = (float*)malloc(sizeinBytes);
  C = (float*)malloc(sizeinBytes);
  D = (float*)malloc(sizeinBytes);
 
  HANDLE_NULL(A);
  HANDLE_NULL(B);
  HANDLE_NULL(C);
  HANDLE_NULL(D);

  // Time
  HANDLE_ERROR( hipEventCreate( &start ) );
  HANDLE_ERROR( hipEventCreate( &stop ) );
  HANDLE_ERROR( hipEventRecord( start, 0 ) );

  // Memory allocation
  HANDLE_ERROR( hipMalloc(&d_A, sizeinBytes) );
  HANDLE_ERROR( hipMalloc(&d_B, sizeinBytes) );
  HANDLE_ERROR( hipMalloc(&d_C, sizeinBytes) );

  // Host initializing
  Initialize(A, N*BLOCK_SIZE, N*BLOCK_SIZE);
  Initialize(B, N*BLOCK_SIZE, N*BLOCK_SIZE);

  // Device initializing
  HANDLE_ERROR( hipMemcpy(d_A, A, sizeinBytes, hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(d_B, B, sizeinBytes, hipMemcpyHostToDevice) );

  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(N, N);

  matmul_kernel<<<dimGrid, dimBlock>>>(d_C, d_A, d_B, N*BLOCK_SIZE);

  HANDLE_ERROR( hipMemcpy(C, d_C, sizeinBytes, hipMemcpyDeviceToHost) );
  
  HANDLE_ERROR( hipEventRecord( stop, 0 ) );
  HANDLE_ERROR( hipEventSynchronize( stop ) );

  float   elapsedTime;
  HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );

  matmul_seq(D, A, B, N*BLOCK_SIZE);

  printf("%s: N = %4d Test = %d Time = %3.1f ms\n", argv[0], N*BLOCK_SIZE, \
         Compare(D, C, N*BLOCK_SIZE, N*BLOCK_SIZE), elapsedTime);

#ifdef DEBUG
  display(A, N*BLOCK_SIZE, N*BLOCK_SIZE);
  display(B, N*BLOCK_SIZE, N*BLOCK_SIZE);
  display(C, N*BLOCK_SIZE, N*BLOCK_SIZE);
#endif

  HANDLE_ERROR( hipEventDestroy( start ) );
  HANDLE_ERROR( hipEventDestroy( stop ) );

  HANDLE_ERROR ( hipFree(d_A) );
  HANDLE_ERROR ( hipFree(d_B) );
  HANDLE_ERROR ( hipFree(d_C) );

  free(A);
  free(B);
  free(C);
  free(D);

  return 0;
}

