#include "hip/hip_runtime.h"
#include <stdio.h>
#include "common.h"
#include "matrix_common.h"

#define BLOCK_SIZE 16
#define DEFAULT_N 1

#define AS(i, j) As[i][j]
#define BS(i, j) Bs[i][j]

////////////////////////////////////////////////////////////////////////////////
//! Matrix multiplication on the device: C = A * B
//! wA is A's width and wB is B's width
////////////////////////////////////////////////////////////////////////////////
__global__ void matrixMul(float* C, float* A, float* B, int N) {
  // Block index
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Index of the first sub-matrix of A processed by the block
  int aBegin = N * BLOCK_SIZE * by;

  // Index of the last sub-matrix of A processed by the block
  int aEnd   = aBegin + N - 1;

  // Step size used to iterate through the sub-matrices of A
  int aStep  = BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
  int bBegin = BLOCK_SIZE * bx;

  // Step size used to iterate through the sub-matrices of B
  int bStep  = BLOCK_SIZE * N;

  // Csub is used to store the element of the block sub-matrix
  // that is computed by the thread
  float Csub = 0;

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix
    As[ty][tx] = A[a + N * ty + tx];
    Bs[ty][tx] = B[b + N * ty + tx];

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
    // #pragma unroll
    for (int k = 0; k < BLOCK_SIZE; ++k)
      Csub += As[ty][k] * Bs[k][tx];

     // Synchronize to make sure that the preceding
     // computation is done before loading two new
     // sub-matrices of A and B in the next iteration
     __syncthreads();
  }

  // Write the block sub-matrix to device memory;
  // each thread writes one element
  int c = N * BLOCK_SIZE * by + BLOCK_SIZE * bx;
  C[c + N * ty + tx] = Csub;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
  int N, sizeinBytes;
  float *A, *B, *C, *D;
  float *d_A, *d_B, *d_C;

  hipEvent_t start, stop;

  /* Command line parameters processing */
  switch(argc) {
    case 1: N = DEFAULT_N;
            break;
    case 2: N = atoi(argv[1]);
            break;
    default:
            printf("\nUse: %s <N>", argv[0]);
            printf("\nN: Dimensión del GRID en bloques de %d)\n", BLOCK_SIZE);
            break;
  }

  sizeinBytes = N*N*BLOCK_SIZE*BLOCK_SIZE*sizeof(float);

  A = (float*)malloc(sizeinBytes);
  B = (float*)malloc(sizeinBytes);
  C = (float*)malloc(sizeinBytes);
  D = (float*)malloc(sizeinBytes);

  HANDLE_NULL(A);
  HANDLE_NULL(B);
  HANDLE_NULL(C);
  HANDLE_NULL(D);

  // Timing
  HANDLE_ERROR( hipEventCreate( &start ) );
  HANDLE_ERROR( hipEventCreate( &stop ) );
  HANDLE_ERROR( hipEventRecord( start, 0 ) );

  // Memory allocation
  HANDLE_ERROR( hipMalloc(&d_A, sizeinBytes) );
  HANDLE_ERROR( hipMalloc(&d_B, sizeinBytes) );
  HANDLE_ERROR( hipMalloc(&d_C, sizeinBytes) );

  // Host initializing
  Initialize(A, N*BLOCK_SIZE, N*BLOCK_SIZE);
  Initialize(B, N*BLOCK_SIZE, N*BLOCK_SIZE);

  // Device initializing
  HANDLE_ERROR( hipMemcpy(d_A, A, sizeinBytes, hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(d_B, B, sizeinBytes, hipMemcpyHostToDevice) );

  // setup execution parameters
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(N, N);

  // execute the kernel
  matrixMul<<< dimGrid, dimBlock >>>(d_C, d_A, d_B, N*BLOCK_SIZE);

  // Results moving
  HANDLE_ERROR( hipMemcpy(C, d_C, sizeinBytes, hipMemcpyDeviceToHost) );

  // Timing
  HANDLE_ERROR( hipEventRecord( stop, 0 ) );
  HANDLE_ERROR( hipEventSynchronize( stop ) );
  float   elapsedTime;
  HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );

  matmul_seq(D, A, B, N*BLOCK_SIZE);

  printf("%s: N = %4d Test = %d Time = %3.1f ms\n", argv[0], N*BLOCK_SIZE, \
         Compare(D, C, N*BLOCK_SIZE, N*BLOCK_SIZE), elapsedTime);

  HANDLE_ERROR( hipEventDestroy( start ) );
  HANDLE_ERROR( hipEventDestroy( stop ) );

  HANDLE_ERROR ( hipFree(d_A) );
  HANDLE_ERROR ( hipFree(d_B) );
  HANDLE_ERROR ( hipFree(d_C) );

  free(A);
  free(B);
  free(C);
  free(D);

  return 0;
}

